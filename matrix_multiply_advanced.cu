#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

// Error checking macro
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Tile size for shared memory optimization
#define TILE_SIZE 32

// Naive matrix multiplication kernel
__global__ void matrixMultiplyNaive(float* A, float* B, float* C, 
                                   int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Tiled matrix multiplication with shared memory
__global__ void matrixMultiplyTiled(float* A, float* B, float* C, 
                                   int M, int N, int K) {
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tile of A into shared memory
        if (row < M && tile * TILE_SIZE + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + tile * TILE_SIZE + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        // Load tile of B into shared memory
        if (tile * TILE_SIZE + threadIdx.y < K && col < N) {
            sB[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Coalesced memory access version
__global__ void matrixMultiplyCoalesced(float* A, float* B, float* C, 
                                       int M, int N, int K) {
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tile of A into shared memory (coalesced access)
        if (row < M && tile * TILE_SIZE + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + tile * TILE_SIZE + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        // Load tile of B into shared memory (coalesced access)
        if (tile * TILE_SIZE + threadIdx.y < K && col < N) {
            sB[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// Warp-level optimization version
__global__ void matrixMultiplyWarpOptimized(float* A, float* B, float* C, 
                                           int M, int N, int K) {
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; tile++) {
        // Load tile of A into shared memory
        if (row < M && tile * TILE_SIZE + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + tile * TILE_SIZE + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        // Load tile of B into shared memory
        if (tile * TILE_SIZE + threadIdx.y < K && col < N) {
            sB[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product with warp-level optimization
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Write result with warp-level optimization
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// CPU reference implementation
void matrixMultiplyCPU(float* A, float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

// Initialize matrix with random values
void initializeMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}

// Print matrix (for small matrices)
void printMatrix(float* matrix, int rows, int cols, const char* name) {
    printf("%s:\n", name);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%6.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Verify results
bool verifyResults(float* cpu_result, float* gpu_result, int size, float tolerance = 1e-5) {
    for (int i = 0; i < size; i++) {
        if (fabs(cpu_result[i] - gpu_result[i]) > tolerance) {
            printf("Mismatch at index %d: CPU=%f, GPU=%f\n", 
                   i, cpu_result[i], gpu_result[i]);
            return false;
        }
    }
    return true;
}

// Time a kernel execution
float timeKernel(void (*kernel)(float*, float*, float*, int, int, int),
                float* d_A, float* d_B, float* d_C, int M, int N, int K,
                dim3 gridDim, dim3 blockDim, const char* kernel_name) {
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    // Warm up
    kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Time the kernel
    CHECK_CUDA(hipEventRecord(start));
    kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    
    float time;
    CHECK_CUDA(hipEventElapsedTime(&time, start, stop));
    
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    
    return time;
}

int main(int argc, char** argv) {
    // Matrix dimensions
    int M = 1024;  // Rows of A
    int N = 1024;  // Columns of B
    int K = 1024;  // Columns of A / Rows of B
    
    // Parse command line arguments
    if (argc >= 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    }
    
    printf("Advanced Matrix Multiplication: A(%dx%d) * B(%dx%d) = C(%dx%d)\n", M, K, K, N, M, N);
    
    // Allocate host memory
    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);
    
    float *h_A = (float*)malloc(sizeA);
    float *h_B = (float*)malloc(sizeB);
    float *h_C = (float*)malloc(sizeC);
    float *h_C_CPU = (float*)malloc(sizeC);
    
    if (!h_A || !h_B || !h_C || !h_C_CPU) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return -1;
    }
    
    // Initialize matrices
    srand(42);  // Fixed seed for reproducibility
    initializeMatrix(h_A, M, K);
    initializeMatrix(h_B, K, N);
    
    // Print small matrices for verification
    if (M <= 8 && N <= 8 && K <= 8) {
        printMatrix(h_A, M, K, "Matrix A");
        printMatrix(h_B, K, N, "Matrix B");
    }
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, sizeA));
    CHECK_CUDA(hipMalloc(&d_B, sizeB));
    CHECK_CUDA(hipMalloc(&d_C, sizeC));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));
    
    // Define block and grid dimensions
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, 
                 (M + blockDim.y - 1) / blockDim.y);
    
    printf("Grid: (%d, %d), Block: (%d, %d)\n", 
           gridDim.x, gridDim.y, blockDim.x, blockDim.y);
    
    // CPU reference computation
    clock_t cpu_start = clock();
    matrixMultiplyCPU(h_A, h_B, h_C_CPU, M, N, K);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000.0;
    
    // Test different kernels
    float naive_time = timeKernel(matrixMultiplyNaive, d_A, d_B, d_C, M, N, K, 
                                 gridDim, blockDim, "Naive");
    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));
    bool naive_correct = verifyResults(h_C_CPU, h_C, M * N);
    
    float tiled_time = timeKernel(matrixMultiplyTiled, d_A, d_B, d_C, M, N, K, 
                                 gridDim, blockDim, "Tiled");
    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));
    bool tiled_correct = verifyResults(h_C_CPU, h_C, M * N);
    
    float coalesced_time = timeKernel(matrixMultiplyCoalesced, d_A, d_B, d_C, M, N, K, 
                                     gridDim, blockDim, "Coalesced");
    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));
    bool coalesced_correct = verifyResults(h_C_CPU, h_C, M * N);
    
    float warp_time = timeKernel(matrixMultiplyWarpOptimized, d_A, d_B, d_C, M, N, K, 
                                gridDim, blockDim, "Warp Optimized");
    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost));
    bool warp_correct = verifyResults(h_C_CPU, h_C, M * N);
    
    // Print results
    printf("\n=== Performance Results ===\n");
    printf("CPU Time: %.3f ms\n", cpu_time);
    printf("GPU Naive Time: %.3f ms (Speedup: %.2fx)\n", 
           naive_time, cpu_time / naive_time);
    printf("GPU Tiled Time: %.3f ms (Speedup: %.2fx)\n", 
           tiled_time, cpu_time / tiled_time);
    printf("GPU Coalesced Time: %.3f ms (Speedup: %.2fx)\n", 
           coalesced_time, cpu_time / coalesced_time);
    printf("GPU Warp Optimized Time: %.3f ms (Speedup: %.2fx)\n", 
           warp_time, cpu_time / warp_time);
    
    printf("\n=== Correctness Results ===\n");
    printf("Naive Kernel Correct: %s\n", naive_correct ? "YES" : "NO");
    printf("Tiled Kernel Correct: %s\n", tiled_correct ? "YES" : "NO");
    printf("Coalesced Kernel Correct: %s\n", coalesced_correct ? "YES" : "NO");
    printf("Warp Optimized Kernel Correct: %s\n", warp_correct ? "YES" : "NO");
    
    // Print small result matrix
    if (M <= 8 && N <= 8) {
        printMatrix(h_C_CPU, M, N, "CPU Result");
        printMatrix(h_C, M, N, "GPU Result (Warp Optimized)");
    }
    
    // Calculate FLOPS
    long long flops = 2LL * M * N * K;
    printf("\n=== FLOPS Analysis ===\n");
    printf("Total FLOPS: %lld\n", flops);
    printf("GPU Naive GFLOPS: %.2f\n", (flops / 1e9) / (naive_time / 1000.0));
    printf("GPU Tiled GFLOPS: %.2f\n", (flops / 1e9) / (tiled_time / 1000.0));
    printf("GPU Coalesced GFLOPS: %.2f\n", (flops / 1e9) / (coalesced_time / 1000.0));
    printf("GPU Warp Optimized GFLOPS: %.2f\n", (flops / 1e9) / (warp_time / 1000.0));
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_CPU);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    
    return 0;
} 